#include "hip/hip_runtime.h"
/* ****************************************************** */
/*    Wilson fermion solver in C language                 */
/*                                                        */
/*    OpenACC benchmark [5 May 0216 H.Matsufuru]          */
/*                                                        */
/*                     Copyright(c) Hideo Matsufuru 2016  */
/* ****************************************************** */

#include "lattice.h"

static real_t u[NDF*NST2*4];
static real_t corr[LT];
#ifdef _PROF
double prof_t[PROF_NUMS];
#endif
MPI_Comm comm_ud, comm_lr;

double dtime()
{
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return ((double)(tv.tv_sec) + (double)(tv.tv_usec) * 1.0e-6);
}

static void create_newcomm(const int pt, const int pz, const int me)
{
  /*
  0 4
  1 5
  2 6
  3 7  PT = 2, PZ = 4 */

  MPI_Comm_split(MPI_COMM_WORLD, me%pz, me/pz, &comm_lr); // color, key
  MPI_Comm_split(MPI_COMM_WORLD, me/pz, me%pz, &comm_ud); // color, key
}

static void uinit(const int me, const int pz, real_t *u)
{
  int Nx2 =  8;
  int Ny2 =  8;
  int Nz2 =  8;
  int Nt2 = 16;
  int Nst2 = Nx2 * Ny2 * Nz2 * Nt2;

  FILE *fp;
  fp = fopen("conf_08080816.txt","r");
  double *ur = (double*)malloc(sizeof(double) * NDF * 4 * Nst2);

  for(int ist = 0; ist < Nst2; ist++){
    for(int idir = 0; idir < 4; idir++){
      for(int idf = 0; idf < NDF; idf++){
	int i = idf + ist*NDF + idir*NDF*Nst2;
	int ret = fscanf(fp, "%lf", &ur[i]);
	if(!ret){
          fprintf(stderr, "Read Error!\n");
          MPI_Finalize();
          exit(0);
	}
      }
    }
  }

  fclose(fp);

  int idir, it, iz, iy, ix;
  for(idir = 0; idir < 4; idir++){
    for(it = 1; it < LT2-1; it++){
      for(iz = 1; iz < LZ2-1; iz++){
	for(iy = 0; iy < NY; iy++){
	  for(ix = 0; ix < NX; ix++){
	    int ist = ix + NX*(iy + NY*(iz + LZ2*it));
	    int ix2 = ix % Nx2;
	    int iy2 = iy % Ny2;
            int iz2 = ((iz-1)+((me%pz)*(LZ2-2))) % Nz2;
            int it2 = ((it-1)+((me/pz)*(LT2-2))) % Nt2;
	    int ist2 = ix2 + Nx2*(iy2 + Ny2*(iz2 + Nz2*it2));
	    for(int idf = 0; idf < NDF; idf++){
	      int i  = idf + NDF*(ist  + idir*NX*NY*LZ2*LT2);
	      int i2 = idf + NDF*(ist2 + idir*Nst2);
	      u[i] = (real_t)ur[i2];
	    }
	  }
	}
      }
    }
  }
  free(ur);
}

__device__ static void setconst(real_t *v, const real_t a)
{
  int i = IDXV(threadIdx.x, blockIdx.x, blockDim.x);
  
  while(i < (LT2-2)*(LZ2-2)*yx_Spinor){
    int t = i / ((LZ2-2)*yx_Spinor);
    int z = (i - t * (LZ2-2)*yx_Spinor)/yx_Spinor;   // (i % ((LZ2-2)*yx_Spinor)) / yx_Spinor;
    int offset = i % yx_Spinor;
    v[(t+1)*LZ2*yx_Spinor + (z+1)*yx_Spinor + offset] = a;
    i += blockDim.x * gridDim.x;
  }
}

__global__ static void set_src(const int me, const int ic, const int id, const int ix, const int iy, const int iz, const int it,
			       real_t *v)
{
  setconst(v, 0.0);
  if(me == 0){ // fix me
    if(threadIdx.x == 0 && blockIdx.x == 0){
      int i = 2*ic + id*NVC + NVC*ND*(ix + iy*NX + (iz+1)*NX*NY + (it+1)*NX*NY*LZ2);
      v[i] = 1.0;
    }
  }
}

static void test_mult(const int me, real_t *u)
{
  int nrepeat = 100;
  real_t *bq2, *xq2;
  HANDLE_ERROR( hipMalloc( (void**)&bq2, NVST2*sizeof(real_t) ) );
  HANDLE_ERROR( hipMalloc( (void**)&xq2, NVST2*sizeof(real_t) ) );
  set_src <<< NUM_GANGS, VECTOR_LENGTH >>> (me, 0, 0, 0, 0, 0, 0, bq2);

  MPI_Barrier(MPI_COMM_WORLD);
  double time0 = dtime();
  for(int i=0; i<nrepeat; i++){
    opr_DdagD_alt(xq2, u, bq2);
    opr_DdagD_alt(bq2, u, xq2);
  }
  MPI_Barrier(MPI_COMM_WORLD);
  double time_tot  = dtime() - time0;
  double fop_mult1 = 2.0 * 1392.0 * (double)(NST);
  double fop_mult  = (double)nrepeat * 2.0 * fop_mult1;

  if(me == 0){
    printf("\nperformance of mult on Host:\n");
    printf("  elapsed time for solver   = %f\n",  time_tot);
    printf("  floating point operations = %f\n",  fop_mult);
    printf("  performance of mult = %f GFlops\n", fop_mult/time_tot * 1.0e-9);
  }

  HANDLE_ERROR( hipFree(bq2) );
  HANDLE_ERROR( hipFree(xq2) );
}

int main(int argc, char *argv[])
{
  real_t enorm = 1.E-16;
  real_t diff;
  int nconv;

  int  namelen, me, nprocs;
  char processor_name[MPI_MAX_PROCESSOR_NAME];

  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
  MPI_Comm_rank(MPI_COMM_WORLD, &me);
  MPI_Get_processor_name(processor_name, &namelen);
  printf("Process %d of %d is on %s\n", me, nprocs, processor_name);

  //  hipSetDevice(me%NGPUS);

  if(me == 0){
    printf("Simple Wilson solver\n\n");
    printf("NX = %3d, NY = %3d, NZ = %3d, NT = %3d\n", NX, NY, NZ, NT);
    printf("LX = %3d, LY = %3d, LZ = %3d, LT = %3d\n", NX, NY, LZ, LT);
    printf("(PT x PZ) = (%d x %d)\n", PT, PZ);
    printf("CKs = %10.6f\n", CKs);
    printf("enorm = %12.4e\n", enorm);
    printf("NUM=%d LEN=%d\n", NUM_GANGS, VECTOR_LENGTH);
  }

  for(int it = 0; it < LT; it++)
    corr[it] = 0.0;
  
  real_t *u_dev, *xq_dev, *bq_dev;
  HANDLE_ERROR( hipMalloc( (void**)&u_dev,  4*LT2*LZ2*yx_Matrix*sizeof(real_t) ) );
  create_newcomm(PT, PZ, me);
  create_cart(PT, PZ, me);

  uinit(me, PZ, u);
  HANDLE_ERROR( hipMemcpy(u_dev, u, 4*LT2*LZ2*yx_Matrix*sizeof(real_t), hipMemcpyHostToDevice) );
  test_mult(me, u_dev);

  HANDLE_ERROR( hipMalloc( (void**)&xq_dev, NVST2*sizeof(real_t) ) );
  HANDLE_ERROR( hipMalloc( (void**)&bq_dev, NVST2*sizeof(real_t) ) );
  if(me == 0){
    printf("Solver:\n");
    printf("  ic  id   nconv      diff\n");
  }
  double time_tot = 0.0;
  double fop_tot  = 0.0;
  
  for(int ic = 0; ic < NCOL; ic++){
    for(int id = 0; id < ND; id++){
      set_src<<< NUM_GANGS,VECTOR_LENGTH >>>(me, ic, id, 0, 0, 0, 0, bq_dev);
      MPI_Barrier(MPI_COMM_WORLD);
      double time0 = dtime();
      solve_CG(enorm, &nconv, &diff, xq_dev, u_dev, bq_dev);
      MPI_Barrier(MPI_COMM_WORLD);
      double time1 = dtime();
      time_tot += time1 - time0;
      
      if(me == 0)
	printf(" %3d %3d  %6d %12.4e\n", ic, id, nconv, diff);

      double fop_mult1 = 2.0 * 1392.0 * (double)(NST);
      double fop_mult  = (double)(nconv+2) * fop_mult1;
      double fop_lin   = (double)(4+(nconv+1)*11) * (double)(NVST);
      fop_tot  += fop_lin + fop_mult;

      norm2_t(corr, xq_dev);
    }
  }

  real_t corr2[NT];

  if(PZ != 1)
    MPI_Allreduce(MPI_IN_PLACE, corr, LT, MPI_DOUBLE, MPI_SUM, comm_ud);

  if(PT != 1)
    MPI_Allgather(corr, LT, MPI_DOUBLE, corr2, LT, MPI_DOUBLE, comm_lr);
  else
    memcpy(corr2, corr, sizeof(real_t)*LT);

  if(me == 0){  
    printf("\nperformance of solver:\n");
    printf("  elapsed time for solver   = %f\n", time_tot);
    printf("  floating point operations = %f\n", fop_tot);
    printf("  performance of solver = %f GFlops\n", fop_tot/time_tot * 1.0e-9);
    printf("\nsolution squared at each time slice:\n");
    for(int it = 0; it < NT; it++)
      printf(" %6d   %16.8e\n", it, corr2[it]);
  }

#ifdef _PROF
  double prof_t_max[PROF_NUMS], prof_t_min[PROF_NUMS], prof_t_ave[PROF_NUMS];
  MPI_Allreduce(prof_t, prof_t_max, PROF_NUMS, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
  MPI_Allreduce(prof_t, prof_t_min, PROF_NUMS, MPI_DOUBLE, MPI_MIN, MPI_COMM_WORLD);
  MPI_Allreduce(prof_t, prof_t_ave, PROF_NUMS, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);

  for(int i=0;i<PROF_NUMS;i++)
    prof_t_ave[i] /= nprocs;

  if(me == 0)
  {
    printf("MAX: PACK %f COMM %f OPR %f COPY %f AXPY %f NORM %f DOT %f SCAL %f\n",
           prof_t_max[PACK], prof_t_max[COMM], prof_t_max[OPR], prof_t_max[COPY],
           prof_t_max[AXPY], prof_t_max[NORM], prof_t_max[DOT], prof_t_max[SCAL]);

    printf("MIN: PACK %f COMM %f OPR %f COPY %f AXPY %f NORM %f DOT %f SCAL %f\n",
           prof_t_min[PACK], prof_t_min[COMM], prof_t_min[OPR], prof_t_min[COPY],
           prof_t_min[AXPY], prof_t_min[NORM], prof_t_min[DOT], prof_t_min[SCAL]);

    printf("AVE: PACK %f COMM %f OPR %f COPY %f AXPY %f NORM %f DOT %f SCAL %f\n",
           prof_t_ave[PACK], prof_t_ave[COMM], prof_t_ave[OPR], prof_t_ave[COPY],
           prof_t_ave[AXPY], prof_t_ave[NORM], prof_t_ave[DOT], prof_t_ave[SCAL]);
  }
#endif
  MPI_Finalize();
  return 0;
}
